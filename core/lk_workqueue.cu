#include "hip/hip_runtime.h"
#include "utils.h"
#include "lk_globals.h"
#include "lk_mailbox.h"
#include "gpu_matmul.h"
#include "lk_utils.h"
#include "lk_workqueue.h"

Task h_queue[WORK_QUEUE_LENGTH];
int h_queueHead, h_taskCounter = 0;

__device__ Task *d_queue;
__device__ int *d_tail, *d_taskCounter;

#define DeviceWriteMyMailboxFrom(_val)  _vcast(from_device[blockIdx.x]) = (_val)

void initQueue() {
	h_queueHead = h_taskCounter = 0;
	hipMalloc(&d_queue, WORK_QUEUE_LENGTH * sizeof(Task));
	hipMalloc(&d_tail, sizeof(int));
	hipMalloc(&d_taskCounter, sizeof(int));

	int zero = 0;

	hipMemcpy(d_queue, h_queue, WORK_QUEUE_LENGTH * sizeof(Task), hipMemcpyHostToDevice);
	hipMemcpy(d_tail, &zero, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_taskCounter, &h_taskCounter, sizeof(int), hipMemcpyHostToDevice);
}


//__device__ const WorkFn lkTasks[] = {naive_wrapper, shared_wrapper};

const char* lkTasksDesc[] = {"naive", "shared_mem"};


__device__ void sleep() {
	return;
}

__device__ bool execute(Task* task) {
	//lkTasks[task->input.fn](task->input.arg, task->res);
	naive_wrapper(task->input.arg, task->res);
	return true;
}

__device__ bool dequeue(volatile mailbox_elem_t * from_device){
	int count = atomicSub(d_taskCounter, 1);

	if(count <= 0) {
		atomicAdd(d_taskCounter, 1);
		sleep();
		return false;
	}

	int tail = atomicAdd(d_tail, 1);
	int idx = tail % WORK_QUEUE_LENGTH;

	execute(&d_queue[idx]);

    DeviceWriteMyMailboxFrom(THREAD_FINISHED);
	return true;
}




bool enqueue(Task task) {
	log("Enqueueing task: %s", lkTasksDesc[task.input.fn]);
	if(h_taskCounter >= WORK_QUEUE_LENGTH) {
		return false;
	}

	int idx = h_queueHead % WORK_QUEUE_LENGTH;
	h_queue[idx] = task;

	hipMemcpy(&(d_queue + idx)->input, &task.input, sizeof(Input), hipMemcpyHostToDevice);

	h_taskCounter++;
	h_queueHead++;
	
	hipMemcpy(d_taskCounter, &h_taskCounter, sizeof(int), hipMemcpyHostToDevice);

	//HostWriteMyMailboxTo(THREAD_WORK);	

	return true;
} 
